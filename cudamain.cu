#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "msglib.h"

#include <iostream>
#include <chrono>
#include <cmath>
#include <thread>
#include <iomanip>
#include <string.h>
#include <cassert>
#include "projetIFT630.h"

#define SHOW_INTERVAL_MS 500
#define BLOCK_SIZE 256
#define SHA_PER_ITERATIONS 8'388'608
#define NUMBLOCKS (SHA_PER_ITERATIONS + BLOCK_SIZE - 1) / BLOCK_SIZE

struct messageBuffer { std::string hash; };

static size_t difficulty = 5;

// Output string by the device read by host
char *g_out = nullptr;
unsigned char *g_hash_out = nullptr;
int *g_found = nullptr;

static uint64_t nonce = 0;

__device__ bool checkZeroPadding(unsigned char* sha, size_t difficulty) {

	for (size_t cur_byte = 0; cur_byte < difficulty / 2; ++cur_byte) {
		if (sha[cur_byte] != 0) {
			return false;
		}
	}

	bool isOdd = difficulty % 2 != 0;
	size_t last_byte_check = static_cast<size_t>(difficulty / 2);
	if (isOdd) {
		if (sha[last_byte_check] > 0x0F || sha[last_byte_check] == 0) {
			return false;
		}
	}
	else if (sha[last_byte_check] < 0x0F) return false;

	return true;
}

// Does the same as sprintf(char*, "%d%s", int, const char*) but a bit faster
__device__ size_t nonce_to_str(uint64_t nonce, unsigned char* out) {
	uint64_t result = nonce;
	uint8_t remainder;
	size_t nonce_size = nonce == 0 ? 1 : floor(log10((double)nonce)) + 1;
	size_t i = nonce_size;
	while (result >= 10) {
		remainder = result % 10;
		result /= 10;
		out[--i] = remainder + '0';
	}

	out[0] = result + '0';
	i = nonce_size;
	out[i] = 0;
	return i;
}


extern __shared__ char array[];
__global__ void sha256_kernel(char* out_input_string_nonce, unsigned char* out_found_hash, int *out_found, const char* in_input_string, size_t in_input_string_size, size_t difficulty, uint64_t nonce_offset) 
{

	// If this is the first thread of the block, init the input string in shared memory
	char* in = (char*) &array[0];
	if (threadIdx.x == 0) 
	{
		memcpy(in, in_input_string, in_input_string_size + 1);
	}

	__syncthreads(); // Ensure the input string has been written in SMEM

	uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t nonce = idx + nonce_offset;

	// The first byte we can write because there is the input string at the begining
	// Respects the memory padding of 8 bit (char).
	size_t const minArray = static_cast<size_t>(ceil((in_input_string_size + 1) / 8.f) * 8);

	uintptr_t sha_addr = threadIdx.x * (64) + minArray;
	uintptr_t nonce_addr = sha_addr + 32;

	unsigned char* sha = (unsigned char*)&array[sha_addr];
	unsigned char* out = (unsigned char*)&array[nonce_addr];
	memset(out, 0, 32);

	size_t size = nonce_to_str(nonce, out);

	assert(size <= 32);

	SHA256_CTX ctx;
	sha256_init(&ctx);
	sha256_update(&ctx, out, size);
	sha256_update(&ctx, (unsigned char *)in, in_input_string_size);
	sha256_final(&ctx, sha);

	if (checkZeroPadding(sha, difficulty) && atomicExch(out_found, 1) == 0) 
{
		memcpy(out_found_hash, sha, 32);
		memcpy(out_input_string_nonce, out, size);
		memcpy(out_input_string_nonce + size, in, in_input_string_size + 1);
	}
}

void pre_sha256() {
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}

// Prints a 32 bytes sha256 to the hexadecimal form filled with zeroes
void print_hash(const unsigned char* sha256) 
{
	std::cout << "Block mined: ";
	for (size_t i = 0; i < 32; ++i) {
		std::cout << std::hex << std::setfill('0') << std::setw(2) << static_cast<int>(sha256[i]);
	}
	std::cout << std::dec << std::endl;
}

void print_state() 
{
	if (*g_found) 
	{
		print_hash(g_hash_out);
	}
}

void addBlock(std::string hash) 
{
	nonce = 0;
	const size_t input_size = hash.size();

	// Input string for the device
	char *d_in = nullptr;

	// Create the input string for the device
	hipMalloc(&d_in, input_size + 1);
	hipMemcpy(d_in, hash.c_str(), input_size + 1, hipMemcpyHostToDevice);

	hipMallocManaged(&g_out, input_size + 32 + 1);
	hipMallocManaged(&g_hash_out, 32);
	hipMallocManaged(&g_found, sizeof(int));
	*g_found = 0;

	pre_sha256();

	size_t dynamic_shared_size = (ceil((input_size + 1) / 8.f) * 8) + (64 * BLOCK_SIZE);

	for (;;) {
		sha256_kernel <<<NUMBLOCKS, BLOCK_SIZE, dynamic_shared_size>>> (g_out, g_hash_out, g_found, d_in, input_size, difficulty, nonce);

		hipError_t err = hipDeviceSynchronize();
		if (err != hipSuccess) {
			throw std::runtime_error("Device error");
		}

		nonce += NUMBLOCKS * BLOCK_SIZE;

		print_state();

		if (*g_found) 
		{
			break;
		}
	}

	hipFree(g_out);
	hipFree(g_hash_out);
	hipFree(g_found);

	hipFree(d_in);

}


int main(int argc, char* argv[]) 
{
	hipSetDevice(0);
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);

	//addBlock depuis argv: HASH DIFFICULTY
	if(argc >= 2)
	{
		std::string h = argv[1];
		difficulty = atoi(argv[2]);
		
		long calcTime;
		calcTime = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
		for(int i =0; i<10; i++)
		{
			std::string newHash(h);
			newHash += i;
			addBlock(newHash);
		}
		std::cout << "Temps pour Cuda: " << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count() - calcTime << " ms." << std::endl;
		return 0;
	}

	std::cout << "Bienvenue dans le projet de blockchain!" << std::endl << std::endl;
	std::cout << "Voici les commandes: " << std::endl;
	std::cout << "1- addBlock from to amount (from, to sont des string. amount est un double) -> ajoute un block au blockchain" << std::endl;
	std::cout << "  -> Exemple: addBlock Mathieu Michael 50.00" << std::endl;
	std::cout << "2- cuda -> part une version automatique CUDA de block mining pour le benchmark" << std::endl;
	std::cout << "3- quit" << std::endl;

	while(true)
	{
		std::string s;
		std::cin >> s;

		if(s.find("addBlock") != std::string::npos)
		{
			std::string in;
			std::cin >> in;
			addBlock(in);
		}
		else if(s.find("cuda") != std::string::npos)
		{
			long calcTime;
			calcTime = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
			std::string in = "\nFROM:Mathieu\nTO:Michael\nAMOUNT:50.0";
			for(int i =0; i<10; i++)
			{
				addBlock(in);
			}
			std::cout << "Temps pour Cuda: " << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count() - calcTime << " ms." << std::endl;


		}
		else if(s.find("quit") != std::string::npos)
		{
			std::cout << "quitter..." << std::endl;
			break;
		}
	}

	hipDeviceReset();


	return 0;
}
